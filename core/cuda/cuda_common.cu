#include "hip/hip_runtime.h"
// Copyright 2009-2022 Intel Corporation
// SPDX-License-Identifier: Apache-2.0

#include "cuda_common.h"

namespace oidn {

  void checkError(hipdnnStatus_t status)
  {
    if (status == HIPDNN_STATUS_SUCCESS)
      return;

    const char* str = hipdnnGetErrorString(status);
    switch (status)
    {
    case HIPDNN_STATUS_ALLOC_FAILED:
      throw Exception(Error::OutOfMemory, str);
    case HIPDNN_STATUS_ARCH_MISMATCH:
    case HIPDNN_STATUS_NOT_SUPPORTED:
      throw Exception(Error::UnsupportedHardware, str);
    default:
      throw Exception(Error::Unknown, str);
    }
  }

  hipdnnDataType_t toCuDNN(DataType dataType)
  {
    switch (dataType)
    {
    case DataType::Float32:
      return HIPDNN_DATA_FLOAT;
    case DataType::Float16:
      return HIPDNN_DATA_HALF;
    case DataType::UInt8:
      return CUDNN_DATA_UINT8;
    default:
      throw std::invalid_argument("unsupported data type");
    }
  }

  hipdnnTensorDescriptor_t toCuDNNTensor(const TensorDesc& td)
  {
    // cuDNN supports tensors with only up to 2G elements
    if (td.getNumElements() > std::numeric_limits<int32_t>::max())
      return nullptr;

    hipdnnTensorFormat_t cuFormat;
    int64_t H, W;

    switch (td.layout)
    {
    case TensorLayout::x:
      cuFormat = HIPDNN_TENSOR_NCHW;
      H = 1;
      W = 1;
      break;
    case TensorLayout::chw:
      cuFormat = HIPDNN_TENSOR_NCHW;
      H = td.dims[1];
      W = td.dims[2];
      break;
    case TensorLayout::hwc:
      cuFormat = HIPDNN_TENSOR_NHWC;
      H = td.dims[1];
      W = td.dims[2];
      break;
    default:
      throw std::invalid_argument("unsupported tensor layout");
    }

    hipdnnTensorDescriptor_t cuDesc;
    checkError(hipdnnCreateTensorDescriptor(&cuDesc));
    checkError(hipdnnSetTensor4dDescriptor(
      cuDesc,
      cuFormat,
      toCuDNN(td.dataType),
      1, int(td.dims[0]), int(H), int(W)));
    return cuDesc;
  }

  hipdnnFilterDescriptor_t toCuDNNFilter(const TensorDesc& td)
  {
    hipdnnTensorFormat_t cuFormat;
    if (td.layout == TensorLayout::oihw)
      cuFormat = HIPDNN_TENSOR_NCHW;
    else if (td.layout == TensorLayout::ohwi)
      cuFormat = HIPDNN_TENSOR_NHWC;
    else
      throw std::invalid_argument("unsupported filter layout");

    hipdnnFilterDescriptor_t cuDesc;
    checkError(hipdnnCreateFilterDescriptor(&cuDesc));
    checkError(hipdnnSetFilter4dDescriptor(
      cuDesc,
      toCuDNN(td.dataType),
      cuFormat,
      int(td.dims[0]), int(td.dims[1]), int(td.dims[2]), int(td.dims[3])));
    return cuDesc;
  }

} // namespace oidn