// Copyright 2009-2021 Intel Corporation
// SPDX-License-Identifier: Apache-2.0

#include "cuda_conv.h"

namespace oidn {

  CUDAConv::CUDAConv(const Ref<CUDADevice>& device, const ConvDesc& desc)
    : CUDAOp(device),
      Conv(desc)
  {
    checkError(hipdnnCreateConvolutionDescriptor(&convDesc));
    checkError(hipdnnSetConvolution2dDescriptor(convDesc,
                                               1,
                                               1,
                                               1,
                                               1,
                                               1,
                                               1,
                                               HIPDNN_CONVOLUTION,
                                               HIPDNN_DATA_HALF));

    // Enable Tensor Core operations
    checkError(hipdnnSetConvolutionMathType(convDesc,
                                           HIPDNN_TENSOR_OP_MATH));

    convAlgo = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM;

    checkError(hipdnnCreateActivationDescriptor(&activationDesc));
    checkError(hipdnnSetActivationDescriptor(activationDesc,
                                            desc.relu ? HIPDNN_ACTIVATION_RELU : HIPDNN_ACTIVATION_PATHTRU,
                                            HIPDNN_PROPAGATE_NAN,
                                            0.));

    srcDesc    = toCuDNNTensor(src->getDesc());
    weightDesc = toCuDNNFilter(weight->getDesc());
    biasDesc   = toCuDNNTensor(bias->getDesc());
    dstDesc    = toCuDNNTensor(dst->getDesc());
  }

  CUDAConv::~CUDAConv()
  {
    checkError(hipdnnDestroyConvolutionDescriptor(convDesc));
    checkError(hipdnnDestroyActivationDescriptor(activationDesc));
    checkError(hipdnnDestroyTensorDescriptor(srcDesc));
    checkError(hipdnnDestroyFilterDescriptor(weightDesc));
    checkError(hipdnnDestroyTensorDescriptor(biasDesc));
    checkError(hipdnnDestroyTensorDescriptor(dstDesc));
  }

  void CUDAConv::run()
  {
    const float alpha1 = 1;
    const float alpha2 = 0;

    checkError(cudnnConvolutionBiasActivationForward(device->getCuDNNHandle(),
                                                     &alpha1,
                                                     srcDesc,
                                                     src->getData(),
                                                     weightDesc,
                                                     weight->getData(),
                                                     convDesc,
                                                     convAlgo,
                                                     scratch ? scratch->getData() : nullptr,
                                                     scratch ? scratch->getByteSize() : 0,
                                                     &alpha2,
                                                     dstDesc,
                                                     dst->getData(),
                                                     biasDesc,
                                                     bias->getData(),
                                                     activationDesc,
                                                     dstDesc,
                                                     dst->getData()));
                                                    

    /*
    checkError(hipdnnConvolutionForward(
      device->getCuDNNHandle(),
      &alpha1,
      srcDesc,
      src->getData(),
      weightDesc,
      weight->getData(),
      convDesc,
      convAlgo,
      nullptr,
      0,
      &alpha2,
      dstDesc,
      dst->getData()));
      */
  }

  size_t CUDAConv::getScratchSize() const
  {
    size_t scratchSize;
    checkError(hipdnnGetConvolutionForwardWorkspaceSize(device->getCuDNNHandle(),
                                                       srcDesc,
                                                       weightDesc,
                                                       convDesc,
                                                       dstDesc,
                                                       convAlgo,
                                                       &scratchSize));
    return scratchSize;
  }

  void CUDAConv::setScratch(const std::shared_ptr<Tensor>& scratch)
  {
    this->scratch = scratch;
  }

} // namespace oidn