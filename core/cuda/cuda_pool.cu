// Copyright 2009-2021 Intel Corporation
// SPDX-License-Identifier: Apache-2.0

#include "cuda_pool.h"

namespace oidn {

  CUDAPool::CUDAPool(const Ref<CUDADevice>& device, const PoolDesc& desc)
    : CUDAOp(device),
      Pool(desc)
  {
    checkError(hipdnnCreatePoolingDescriptor(&poolDesc));
    checkError(hipdnnSetPooling2dDescriptor(poolDesc,
                                           HIPDNN_POOLING_MAX,
                                           HIPDNN_PROPAGATE_NAN,
                                           2,
                                           2,
                                           0,
                                           0,
                                           2,
                                           2));

    srcDesc = toCuDNNTensor(src->getDesc());
    dstDesc = toCuDNNTensor(dst->getDesc());
  }

  CUDAPool::~CUDAPool()
  {
    checkError(hipdnnDestroyPoolingDescriptor(poolDesc));
    checkError(hipdnnDestroyTensorDescriptor(srcDesc));
    checkError(hipdnnDestroyTensorDescriptor(dstDesc));
  }

  void CUDAPool::run()
  {
    const float alpha = 1;
    const float beta  = 0;
    checkError(hipdnnPoolingForward(device->getCuDNNHandle(),
                                  poolDesc,
                                  &alpha,
                                  srcDesc,
                                  src->getData(),
                                  &beta,
                                  dstDesc,
                                  dst->getData()));
  }

} // namespace oidn