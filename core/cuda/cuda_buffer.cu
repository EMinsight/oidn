// Copyright 2009-2021 Intel Corporation
// SPDX-License-Identifier: Apache-2.0

#include "cuda_buffer.h"

namespace oidn {

  void* CUDABufferAllocator::allocate(const Ref<CUDADevice>& device, size_t size, MemoryKind kind)
  {
    void* ptr;

    switch (kind)
    {
    case MemoryKind::Host:
      checkError(hipHostMalloc(&ptr, size));
      return ptr;

    case MemoryKind::Device:
      checkError(hipMalloc(&ptr, size));
      return ptr;

    case MemoryKind::Shared:
      checkError(hipMallocManaged(&ptr, size));
      return ptr;

    default:
      throw Exception(Error::InvalidArgument, "invalid CUDA buffer type");
    }
  }

  void CUDABufferAllocator::deallocate(const Ref<CUDADevice>& device, void* ptr, MemoryKind kind)
  {
    if (kind == MemoryKind::Host)
      checkError(hipHostFree(ptr));
    else
      checkError(hipFree(ptr));
  }

} // namespace oidn
