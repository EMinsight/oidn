// Copyright 2009-2023 Intel Corporation
// SPDX-License-Identifier: Apache-2.0

#include "cuda_device.h"
#include "cuda_engine.h"

OIDN_NAMESPACE_BEGIN

  void checkError(hipError_t error)
  {
    if (error == hipSuccess)
      return;

    const char* str = hipGetErrorString(error);
    switch (error)
    {
    case hipErrorOutOfMemory:
      throw Exception(Error::OutOfMemory, str);
    case hipErrorNoDevice:
    case hipErrorInvalidConfiguration:
    case hipErrorNotSupported:
      throw Exception(Error::UnsupportedHardware, str);
    default:
      throw Exception(Error::Unknown, str);
    }
  }

  bool CUDADevice::isSupported()
  {
    int deviceId = 0;
    if (hipGetDevice(&deviceId) != hipSuccess)
      return false;
    hipDeviceProp_t prop;
    if (hipGetDeviceProperties(&prop, deviceId) != hipSuccess)
      return false;
    const int smArch = prop.major * 10 + prop.minor;
    return smArch >= minSMArch && smArch <= maxSMArch &&
           prop.unifiedAddressing && prop.managedMemory;
  }

  CUDADevice::CUDADevice(int deviceId, hipStream_t stream)
    : deviceId(deviceId),
      stream(stream)
  {
    if (deviceId < 0)
      checkError(hipGetDevice(&this->deviceId));
  }

  CUDADevice::~CUDADevice()
  {
    // Make sure to free up all resources inside a begin/end block
    begin();
    engine = nullptr;
    end();
  }

  void CUDADevice::begin()
  {
    assert(prevDeviceId < 0);

    // Save the current CUDA device
    checkError(hipGetDevice(&prevDeviceId));

    // Set the current CUDA device
    if (deviceId != prevDeviceId)
      checkError(hipSetDevice(deviceId));
  }

  void CUDADevice::end()
  {
    assert(prevDeviceId >= 0);

    // Restore the previous CUDA device
    if (deviceId != prevDeviceId)
      checkError(hipSetDevice(prevDeviceId));
    prevDeviceId = -1;
  }

  void CUDADevice::init()
  {
    hipDeviceProp_t prop;
    checkError(hipGetDeviceProperties(&prop, deviceId));
    maxWorkGroupSize = prop.maxThreadsPerBlock;
    smArch = prop.major * 10 + prop.minor;

    if (isVerbose())
    {
      std::cout << "  Device    : " << prop.name << std::endl;
      std::cout << "    Arch    : SM " << prop.major << "." << prop.minor << std::endl;
      std::cout << "    SMs     : " << prop.multiProcessorCount << std::endl;
    }

    // Check required hardware features
    if (smArch < minSMArch || smArch > maxSMArch)
      throw Exception(Error::UnsupportedHardware, "device has unsupported compute capability");
    if (!prop.unifiedAddressing)
      throw Exception(Error::UnsupportedHardware, "device does not support unified addressing");
    if (!prop.managedMemory)
      throw Exception(Error::UnsupportedHardware, "device does not support managed memory");

    tensorDataType = DataType::Float16;
    tensorLayout   = TensorLayout::hwc;
    weightLayout   = TensorLayout::ohwi;
    tensorBlockC   = 8; // required by Tensor Core operations

#if defined(_WIN32)
    externalMemoryTypes = ExternalMemoryTypeFlag::OpaqueWin32 |
                          ExternalMemoryTypeFlag::OpaqueWin32KMT |
                          ExternalMemoryTypeFlag::D3D11Texture |
                          ExternalMemoryTypeFlag::D3D11TextureKMT |
                          ExternalMemoryTypeFlag::D3D11Resource |
                          ExternalMemoryTypeFlag::D3D11ResourceKMT |
                          ExternalMemoryTypeFlag::D3D12Heap |
                          ExternalMemoryTypeFlag::D3D12Resource;
#else
    externalMemoryTypes = ExternalMemoryTypeFlag::OpaqueFD;
#endif

    engine = makeRef<CUDAEngine>(this, stream);
  }

  Storage CUDADevice::getPointerStorage(const void* ptr)
  {
    hipPointerAttribute_t attrib;
    if (hipPointerGetAttributes(&attrib, ptr) != hipSuccess)
      return Storage::Undefined;

    switch (attrib.type)
    {
    case hipMemoryTypeHost:
      return Storage::Host;
    case hipMemoryTypeDevice:
      return Storage::Device;
    case hipMemoryTypeManaged:
      return Storage::Managed;
    default:
      return Storage::Undefined;
    }
  }

  void CUDADevice::wait()
  {
    engine->wait();
  }

OIDN_NAMESPACE_END